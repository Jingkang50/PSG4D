#include "hip/hip_runtime.h"
#include "otsu.h"
#include "../cuda_utils.h"

__global__ void get_hist_cuda_(int* input, int* hist, int step, int pts_num){
    const Int inst = blockIdx.x; 
    for(Int idx = threadIdx.x; idx<pts_num ; idx += blockDim.x){
        int thre = input[inst * pts_num + idx];
        hist[blockIdx.x * step + thre] += 1;
    }
}


void get_hist_cuda(int* input, int* hist, int step, int inst_num, int pts_num){
    hipError_t err;

    dim3 blocks(inst_num);
    dim3 threads(THREADS_PER_BLOCK);

    int* p_hist;

    hipMalloc((void**)&p_hist, inst_num*step*sizeof(Int));
    hipMemcpy(p_hist, hist, inst_num*step*sizeof(Int), hipMemcpyHostToDevice);

    get_hist_cuda_<<<blocks, threads>>>(input, p_hist, step, pts_num);

    hipMemcpy(hist, p_hist, inst_num*step*sizeof(Int), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}